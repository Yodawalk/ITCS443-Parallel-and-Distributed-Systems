
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 256

__global__ void vecAdd(int *A) {
	int i = threadIdx.x;
	A[i]=A[i]+1;
}

int main (int argc, char *argv[]){
	int i;
	int size = N*sizeof(int);
	int a[N],*devA;

	for (i=0; i< N; i++){
		a[i] = i;
	}
	hipMalloc( (void**)&devA,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

	vecAdd<<<1, N>>>(devA);

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipFree(devA);

	for (i=0; i < N; i++) {
		printf("%d ",a[i]);
	}
	printf("\n");

}
