
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 1000

__global__ void vecAdd(int *A) {
	int i = threadIdx.x;
	A[i]=A[i]+1;
}

int main (int argc, char *argv[]){
	int i;
	int size = N*sizeof(int);
	int a[N],*devA;

	for (i=0; i< N; i++){
		a[i] = i;
	}
	hipMalloc( (void**)&devA,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

	vecAdd<<<1, 256>>>(devA);

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipFree(devA);

	for (i=0; i < N; i++) {
		printf("%d ",a[i]);
	}
	printf("\n");

}
