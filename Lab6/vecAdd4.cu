
#include <hip/hip_runtime.h>
#include <stdio.h>

#define T 1024
#define n 240

__global__ void vecAdd(int *A, int *B, int *C) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n)C[i] = A[i] + B[i];
}

int main (int argc, char *argv[]){
	int i;
	int size = n*sizeof(int);
	int a[n], b[n], c[n], *devA, *devB, *devC;

	for (i=0; i< n; i++){
		a[i] = 1; b[i] =2;
	}
	hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMalloc( (void**)&devC,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

	int blocks = (n + T - 1) /T;
	vecAdd<<<blocks, T>>>(devA, devB, devC);

	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	for (i=0; i < n; i++) {
		printf("%d ",c[i]);
	}
	printf("\n");

}
