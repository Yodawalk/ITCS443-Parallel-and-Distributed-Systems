
#include <hip/hip_runtime.h>
#include <stdio.h>
#define Width 31

#define TITE_WIDTH 16

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Block ID X : %d and  Block ID Y: %d\n", blockIdx.x,blockIdx.y);
	float Pvalue = 0;
	if(row < Width || col < Width){
	for(int k=0;k<ncols;k++){
		float Melement = Md[row*ncols+k];
		float Nelement = Nd[k*ncols+col];
		Pvalue += Melement * Nelement;
	}
}
	
	Pd[row*ncols+col] = Pvalue;
}

int main (int argc, char *argv[]){
	int i,j;
	int size = Width * Width * sizeof(float);
	float M[Width][Width], N[Width][Width], P[Width][Width];
	float* Md, *Nd, *Pd;
	
	for(i=0;i<Width;i++){
		for(j=0;j<Width;j++){
			M[i][j] = 1;
			N[i][j] = 2;
		}
	}
	hipMalloc( (void**)&Md, size);
	hipMalloc( (void**)&Nd, size);
	hipMalloc( (void**)&Pd, size);
	
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice);
	hipMemcpy( Nd, N, size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(TITE_WIDTH, TITE_WIDTH);
	dim3 dimGrid((Width+TITE_WIDTH-1)/TITE_WIDTH,(Width+TITE_WIDTH-1)/TITE_WIDTH);
	
	MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);
	
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	printf("\n================================\n");
	for(i=0;i<Width;i++){
		for(j=0;j<Width;j++){
			printf("%.2f ", P[i][j]);
		}
	}
}
