
#include <hip/hip_runtime.h>
#include <stdio.h>

#define T 64
#define n 256

__global__ void vecAdd(int *A, int *B, int *C) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	C[i] = A[i];
}

int main (int argc, char *argv[]){
	int i;
	int size = n*sizeof(int);
	int a[n], b[n], c[n], *devA, *devB, *devC;

	for (i=0; i< n; i++){
		a[i] = i;	}
	hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMalloc( (void**)&devC,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

	
	vecAdd<<<n/T, T>>>(devA, devB, devC);

	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	for (i=0; i < n; i++) {
		printf("%d ",c[i]);
	}
	printf("\n");

}
