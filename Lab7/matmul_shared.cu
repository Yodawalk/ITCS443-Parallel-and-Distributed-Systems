
#include <hip/hip_runtime.h>
#include<stdio.h>

#define Width 32    // size of Width x Width matrix
#define TILE_WIDTH 16

__global__ void matrixMul(float* A, float* B, float* C, int width)
{
    __shared__ float As[TILE_WIDTH] [TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH] [TILE_WIDTH];
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float c_val = 0.0f;for(int i = 0; i < width/TILE_WIDTH; i++)
    {
        As[threadIdx.y][threadIdx.x] = A[row * width + (i * TILE_WIDTH + threadIdx.x)];
        Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * width + col ];
        __syncthreads();
        for(int k = 0; k < TILE_WIDTH; k++)
        c_val += As[threadIdx.y][k] * Bs[k][threadIdx.x];__syncthreads();
    }
    C[row * width + col] = c_val;
}

int main (int argc, char *argv[] ) {
    const int n = 16;
    int i,j;
    int size = Width * Width * sizeof(float);
    float M[Width][Width],N[Width][Width],P[Width][Width];
    float *Md, *Nd, *Pd;

    for (i=0; i < Width; i++) {
        for (j=0; j < Width; j++) {
            M[i][j] = 1; N[i][j] = 2;
        }
    }

    hipMalloc( (void**)&Md, size);
    hipMalloc( (void**)&Nd, size);
    hipMalloc( (void**)&Pd, size);

    hipMemcpy( Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy( Nd, N, size, hipMemcpyHostToDevice);

    // Setup the execution configuration
    dim3 dimBlock(n, n);
    dim3 dimGrid(Width/n, Width/n);

    // Launch the device computation threads!
    matrixMul<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

    // Read P from the device
    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    // Free device matrices
    hipFree(Md); hipFree(Nd); hipFree(Pd);

    for (i=0; i < Width; i++) {
        for (j=0; j< Width; j++) {
            printf("%.2f ",P[i][j]);
        }
        printf("\n");
    }
}
